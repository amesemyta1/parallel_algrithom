#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include "hip/hip_runtime.h"

#define matrix_size 1000
#define grid_size matrix_size
#define block_size 256
#define matrix_size_2 matrix_size*matrix_size

int *A, *B, *C;


//读B矩阵（B按行存）
void readB(){
      FILE* File_B = fopen("b.txt", "r"); 
      int *temp=B;
      int *up=(int *)(B+matrix_size_2);
      while(temp<up){
         fscanf(File_B, "%d ", temp);
         temp++;
      } 
      fclose(File_B);
}

//读A转置矩阵(A按行存)
void readA(){
      FILE* File_A = fopen("a.txt", "r"); 
      int *temp=A;
      int *up=(int *)(A+matrix_size_2);
      while(temp<up){
         fscanf(File_A, "%d ", temp);
         temp++;
      } 
      fclose(File_A);
}

//输出矩阵C
void writeC(){
      FILE* File_C = fopen ("c.txt", "w" );
      int t, row, col;
      for(row=0; row<matrix_size; row++){
         t=row*matrix_size;
         for(col=0; col<matrix_size; col++){
            fprintf(File_C, "%d ", C[t+col]);
         }
         fprintf(File_C, " \n");
      }
      fclose(File_C);
}



//cuda矩阵乘法
__global__ void multiply(int *D_A, int *D_B, int *D_C){
  int t1= threadIdx.x;
  int t2= blockIdx.x;
  int t3= blockDim.x;
  int t4= t2 * matrix_size;
  int i, temp, j;
  __shared__ int shared_A[1000];
  for(i=t1; i<matrix_size; i+=t3){
     shared_A[i]=D_A[t4+i];
  }
  __syncthreads();

  for(i=t1; i<matrix_size; i+=t3){
     temp=0;
     for(j=0; j<matrix_size; j++){
        temp+=shared_A[j]*D_B[j*matrix_size+i];
     }
     D_C[t4+i]=temp;
  }
  
}


int main()
{
   hipEvent_t event1, event2;
   hipEventCreate(&event1);
   hipEventCreate(&event2);
   
   hipEventRecord(event1, 0 );
   int thread;

   //申请分配设备和主机内存
   int size=sizeof(int)*matrix_size_2;
   int *D_A, *D_B, *D_C;
   A = (int *)malloc(size);
   B = (int *)malloc(size);
   C = (int *)malloc(size);
   hipMalloc((void **)&D_A, size);
   hipMalloc((void **)&D_B, size);
   hipMalloc((void **)&D_C, size);
  
   #ifdef _OPENMP
      printf("paralell read file\n");
      //并行读入矩阵A，B（并把B转置）
      #pragma omp parallel for
      for(thread=0; thread<2; thread++)
         if(thread==0)
           readA();
         else
           readB();
   #else
      printf("serial read file\n");
      readA();
      readB();
   #endif

   //向GPU上传输数据
   hipMemcpy(D_A, A, size, hipMemcpyHostToDevice);
   hipMemcpy(D_B, B, size, hipMemcpyHostToDevice);

   dim3 block(block_size, 1, 1), grid(grid_size, 1, 1);
   
   multiply<<<grid, block>>>(D_A, D_B, D_C);

   //将结果传输会主机
   hipMemcpy(C, D_C, size, hipMemcpyDeviceToHost);

   //输出矩阵C
   writeC();
   

   float time;
   
   hipEventRecord(event2, 0 );
   hipEventSynchronize(event2);
   hipEventElapsedTime(&time, event1, event2);
   hipEventDestroy(event1);
   hipEventDestroy(event2);
   //printf("run time:\t%f s\n",time);

   //释放空间
   free(A);
   free(B);
   free(C);
   hipFree(D_A);
   hipFree(D_B);
   hipFree(D_C);

   return 0;
}

